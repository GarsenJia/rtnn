#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <cuda/helpers.h>
#include <hip/hip_vector_types.h>
#include <optix_device.h>

#include "optixRangeSearch.h"
#include "helpers.h"

extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __raygen__knn()
{
    const uint3 idx = optixGetLaunchIndex();
    unsigned int rayIdx = idx.x;

    unsigned int queryIdx;
    if (params.d_r2q_map == nullptr)
      queryIdx = rayIdx;
    else
      queryIdx = params.d_r2q_map[rayIdx];

    float3 ray_origin = params.queries[queryIdx];
    float3 ray_direction = normalize(make_float3(1, 0, 0));

    unsigned int id = 0;
    const float tmin = 0.f;
    const float tmax = 1.e-16f;

    optixTrace(
        params.handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_NONE,
        //OPTIX_RAY_FLAG_DISABLE_ANYHIT |
        //OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
        RAY_TYPE_RADIANCE,
        1,
        RAY_TYPE_RADIANCE,
        reinterpret_cast<unsigned int&>(queryIdx),
        reinterpret_cast<unsigned int&>(id)
    );

    if (params.d_r2q_map != nullptr || params.limit != 1)
      params.frame_buffer[queryIdx] = id;
}
