#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <optix.h>

#include "optixRangeSearch.h"
#include "helpers.h"

extern "C" {
__constant__ Params params;
}

extern "C" __device__ void intersect_sphere()
{
    unsigned int primIdx = optixGetPrimitiveIndex();
    const float3 center = params.points[primIdx];

    const float3  ray_orig = optixGetWorldRayOrigin();

    bool intersect = false;
    float3 topRight = center + params.radius;
    float3 bottomLeft = center - params.radius;
    if ((ray_orig > bottomLeft) && (ray_orig < topRight))
      intersect = true;

    if (intersect) {
      unsigned int id = optixGetPayload_1();
      if (id < params.limit) {
        unsigned int queryIdx = optixGetPayload_0();
        unsigned int primIdx = optixGetPrimitiveIndex();
        params.frame_buffer[queryIdx * params.limit + id] = primIdx;
        if (id + 1 == params.limit)
          optixReportIntersection( 0, 0 );
        else optixSetPayload_1( id+1 );
      }
    }
}

extern "C" __global__ void __intersection__sphere_radius()
{
  // The IS program will be called if the ray origin is within a primitive's
  // bbox (even if the actual intersections are beyond the tmin and tmax).

  bool isApprox = params.isApprox;

  if (isApprox) {
    unsigned int id = optixGetPayload_1();
    if (id < params.limit) {
      unsigned int queryIdx = optixGetPayload_0();
      unsigned int primIdx = optixGetPrimitiveIndex();
      params.frame_buffer[queryIdx * params.limit + id] = primIdx;
      if (id + 1 == params.limit)
        optixReportIntersection( 0, 0 );
      else optixSetPayload_1( id+1 );
    }
  } else {
    intersect_sphere();
  }
}

extern "C" __global__ void __anyhit__terminateRay()
{
  optixTerminateRay();
}

