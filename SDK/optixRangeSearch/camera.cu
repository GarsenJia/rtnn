#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <hip/hip_vector_types.h>
#include <optix_device.h>
#include "optixRangeSearch.h"
#include "random.h"
#include "helpers.h"
#include <cuda/helpers.h>

extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __raygen__knn()
{
    const uint3 idx = optixGetLaunchIndex();
    unsigned int rayIdx = idx.x;

    // if d_r2q_map is null, it could be 1) an unsorted run, 2) an initial run
    // for sorting, or 3) a sorted run with queries pre-gathered. Either case,
    // we directly map rays to queries.

    unsigned int queryIdx;
    if (params.d_r2q_map == nullptr)
      queryIdx = rayIdx;
    else
      queryIdx = params.d_r2q_map[rayIdx];

    float3 ray_origin = params.queries[queryIdx];
    float3 ray_direction = normalize(make_float3(1, 0, 0));

    unsigned int id = 0;
    const float tmin = 0.f;
    const float tmax = 1.e-16f;

    // TODO: change this to max float. when this get re-casted to float, it will be a big float.
    float a0_key = 10e6;
    unsigned int a0_id = 0xFFFFFFFF;
    float a1_key = 0.f;
    unsigned int a1_id = 0xFFFFFFFF;
    float a2_key = 0.f;
    unsigned int a2_id = 0xFFFFFFFF;

    optixTrace(
        params.handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_DISABLE_ANYHIT |
        OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
        RAY_TYPE_RADIANCE,
        1,
        RAY_TYPE_RADIANCE,
        reinterpret_cast<unsigned int&>(queryIdx),
        reinterpret_cast<unsigned int&>(a0_key),
        reinterpret_cast<unsigned int&>(a0_id),
        reinterpret_cast<unsigned int&>(a1_key),
        reinterpret_cast<unsigned int&>(a1_id),
        reinterpret_cast<unsigned int&>(a2_key),
        reinterpret_cast<unsigned int&>(a2_id)
    );

    params.frame_buffer[queryIdx * params.knn] = a0_id;
    params.frame_buffer[queryIdx * params.knn + 1] = a1_id;
    params.frame_buffer[queryIdx * params.knn + 2] = a2_id;
}

extern "C" __global__ void __raygen__pinhole_camera()
{
    const uint3 idx = optixGetLaunchIndex();
    unsigned int rayIdx = idx.x;

    // if d_r2q_map is null, it could be 1) an unsorted run, 2) an initial run
    // for sorting, or 3) a sorted run with queries pre-gathered. Either case,
    // we directly map rays to queries.

    unsigned int queryIdx;
    if (params.d_r2q_map == nullptr)
      queryIdx = rayIdx;
    else
      queryIdx = params.d_r2q_map[rayIdx];

    float3 ray_origin = params.queries[queryIdx];
    float3 ray_direction = normalize(make_float3(1, 0, 0));

    unsigned int id = 0;
    const float tmin = 0.f;
    const float tmax = 1.e-16f;

    optixTrace(
        params.handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_NONE,
        //OPTIX_RAY_FLAG_DISABLE_ANYHIT |
        //OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
        RAY_TYPE_RADIANCE,
        1,
        RAY_TYPE_RADIANCE,
        reinterpret_cast<unsigned int&>(queryIdx),
        reinterpret_cast<unsigned int&>(id)
    );
    //params.frame_buffer[queryIdx * params.knn] = queryIdx;
    //params.frame_buffer[queryIdx * params.knn+1] = ray_origin.x;
    //params.frame_buffer[queryIdx * params.knn+2] = ray_origin.y;
    //params.frame_buffer[queryIdx * params.knn+3] = ray_origin.z;
}
