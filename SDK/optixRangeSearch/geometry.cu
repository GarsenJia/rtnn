#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <optix.h>

#include "optixRangeSearch.h"
#include "random.h"
#include "helpers.h"

extern "C" {
__constant__ Params params;
}

extern "C" __device__ void intersect_sphere()
{
    // This is called when a ray-bbox intersection is found, but we still can't
    // be sure that the point is within the sphere. It's possible that the
    // point is within the bbox but no within the sphere, and it's also
    // possible that the point is just outside of the bbox and just intersects
    // with the bbox. Note that it's wasteful to do a ray-sphere intersection
    // test and use the intersected Ts to decide whethere a point is inside the
    // sphere or not
    // (https://www.scratchapixel.com/lessons/3d-basic-rendering/minimal-ray-tracer-rendering-simple-shapes/ray-sphere-intersection).

    unsigned int primIdx = optixGetPrimitiveIndex();
    const float3 center = params.points[primIdx];

    const float3  ray_orig = optixGetWorldRayOrigin();

    float3 O = ray_orig - center;

    if (dot(O, O) < params.radius * params.radius) {
      //unsigned int id = optixGetPayload_1();
      //optixSetPayload_1( id+1 );

      unsigned int id = optixGetPayload_1();
      if (id < params.limit) {
        unsigned int queryIdx = optixGetPayload_0();
        unsigned int primIdx = optixGetPrimitiveIndex();
        params.frame_buffer[queryIdx * params.limit + id] = primIdx;
        if (id + 1 == params.limit)
          optixReportIntersection( 0, 0 );
        else optixSetPayload_1( id+1 );
      }
    }
}

extern "C" __global__ void __intersection__sphere_knn()
{
  // The IS program will be called if the ray origin is within a primitive's
  // bbox (even if the actual intersections are beyond the tmin and tmax).

  bool isApprox = false;

  // if d_r2q_map is null and limit is 1, this is the initial run for sorting
  if (params.d_r2q_map == nullptr && params.limit == 1) isApprox = true;

  unsigned int queryIdx = optixGetPayload_0();
  unsigned int primIdx = optixGetPrimitiveIndex();
  if (isApprox) {
    params.frame_buffer[queryIdx * params.limit] = primIdx;
    optixReportIntersection( 0, 0 );
  } else {
    const float3 center = params.points[primIdx];

    const float3  ray_orig = optixGetWorldRayOrigin();
    float3 O = ray_orig - center;
    float distSquared = dot(O, O);

    if ((distSquared > 0) && (distSquared < params.radius * params.radius)) {
      unsigned int t = optixGetPayload_1();
      float a0_key = reinterpret_cast<float&>(t); // a0 always stores the max in the heap
      if (distSquared < a0_key) {
        a0_key = distSquared;
        float a0_id = primIdx;

        t = optixGetPayload_3();
        float a1_key = reinterpret_cast<float&>(t);
        unsigned int a1_id = optixGetPayload_4();
        t = optixGetPayload_5();
        float a2_key = reinterpret_cast<float&>(t);
        unsigned int a2_id = optixGetPayload_6();

        float t_key;
        unsigned int t_id;
        if (a1_key > a0_key) {
          t_key = a0_key; a0_key = a1_key; a1_key = t_key;
          t_id = a0_id; a0_id = a1_id; a1_id = t_id;
        }
        if (a2_key > a0_key) {
          t_key = a0_key; a0_key = a2_key; a2_key = t_key;
          t_id = a0_id; a0_id = a2_id; a2_id = t_id;
        }

        optixSetPayload_1(reinterpret_cast<unsigned int&>(a0_key));
        optixSetPayload_2(a0_id);
        optixSetPayload_3(reinterpret_cast<unsigned int&>(a1_key));
        optixSetPayload_4(a1_id);
        optixSetPayload_5(reinterpret_cast<unsigned int&>(a2_key));
        optixSetPayload_6(a2_id);
      }
    }
  }
}

extern "C" __global__ void __intersection__sphere()
{
  // The IS program will be called if the ray origin is within a primitive's
  // bbox (even if the actual intersections are beyond the tmin and tmax).

  bool isApprox = false;

  // if d_r2q_map is null and limit is 1, this is the initial run for sorting
  if (params.d_r2q_map == nullptr && params.limit == 1) isApprox = true;

  if (isApprox) {
    unsigned int id = optixGetPayload_1();
    if (id < params.limit) {
      unsigned int queryIdx = optixGetPayload_0();
      unsigned int primIdx = optixGetPrimitiveIndex();
      params.frame_buffer[queryIdx * params.limit + id] = primIdx;
      if (id + 1 == params.limit)
        optixReportIntersection( 0, 0 );
      else optixSetPayload_1( id+1 );
    }
  } else {
    intersect_sphere();
  }
}

extern "C" __global__ void __anyhit__terminateRay()
{
  optixTerminateRay();
}

