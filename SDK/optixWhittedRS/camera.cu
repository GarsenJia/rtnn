#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <hip/hip_vector_types.h>
#include <optix_device.h>
#include "optixWhitted.h"
#include "random.h"
#include "helpers.h"
#include <cuda/helpers.h>

extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __raygen__pinhole_camera()
{
    const uint3 idx = optixGetLaunchIndex();
    //const uint3 dim = optixGetLaunchDimensions();

    const GeomData* geom = (GeomData*) optixGetSbtDataPointer();
    //const CameraData* camera = (CameraData*) optixGetSbtDataPointer();

    //const unsigned int image_index = params.width * idx.y + idx.x;
    //unsigned int image_index = params.width * params.numPrims * idx.y + idx.x * params.numPrims;
    unsigned int rayIdx = idx.x;

    // calculate d by transforming <0, 0> from the top-left corner to the center of the image
    //float2 d = make_float2(idx.x, idx.y) / make_float2(params.width, params.height) * 2.f - 1.f;
    //float3 ray_origin = camera->eye;
    //float3 ray_direction = normalize(d.x*camera->U + d.y*camera->V + camera->W);

    float3 ray_origin = geom->spheres[rayIdx];
    float3 ray_direction = normalize(make_float3(1, 1, 1));

    unsigned int id = 0;

    optixTrace(
        params.handle,
        ray_origin,
        ray_direction,
        params.scene_epsilon,
        //1e16f,
        1e-2f,
        0.0f,
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_NONE,
        RAY_TYPE_RADIANCE,
        RAY_TYPE_COUNT,
        RAY_TYPE_RADIANCE,
        reinterpret_cast<unsigned int&>(rayIdx),
        reinterpret_cast<unsigned int&>(id)
    );
}
